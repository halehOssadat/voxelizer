#include "hip/hip_runtime.h"
#include "includes/CompFab.h"
#include "math.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "includes/cuda_math.h"

#include <iostream>
#include <string>
#include <sstream>
#include "stdio.h"
#include <vector>

#include "includes/vox3D.cuh"
#include "includes/square2D.cuh"



// check cuda calls for errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// set up random seed buffer
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

// -------------------------------3D voxelizer -------------------------------
__global__ void voxelize_kernel_3D(vox3D voxelizer, hiprandState* globalState)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	int3 Index;
	Index.x = xIndex;
	Index.y = yIndex;
	Index.z = zIndex;

	if (voxelizer.samples > 0) {
		voxelizer.voxelize_kernel_open_mesh(globalState, Index);
	} else {
		voxelizer.voxelize_kernel(Index);
	}


}

// voxelize the given mesh with the given resolution and dimensions
void kernel_wrapper_3D(int samples, int w, int h, int d, int nodeN, CompFab::VoxelGrid *g_voxelGrid, std::vector<CompFab::Triangle> triangles, bool double_thick)
{
	
	int blocksInX = (w+nodeN-1)/nodeN;
	int blocksInY = (h+nodeN-1)/nodeN;
	int blocksInZ = (d+nodeN-1)/nodeN;

	dim3 Dg(blocksInX, blocksInY, blocksInZ);
	dim3 Db(nodeN, nodeN, nodeN);

	hiprandState* devStates;
	if (samples > 0) {
		// set up random numbers
		dim3 tpb(RANDOM_SEEDS,1,1);
	    hipMalloc ( &devStates, RANDOM_SEEDS*sizeof( hiprandState ) );
	    // setup seeds
	    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );
	}
	
	// set up boolean array on the GPU
	bool *gpu_inside_array;
	gpuErrchk( hipMalloc( (void **)&gpu_inside_array, sizeof(bool) * w * h * d ) );
	gpuErrchk( hipMemcpy( gpu_inside_array, g_voxelGrid->m_insideArray, sizeof(bool) * w * h * d, hipMemcpyHostToDevice ) );

	// set up triangle array on the GPU
	CompFab::Triangle* triangle_array = &triangles[0];
	CompFab::Triangle* gpu_triangle_array;
	gpuErrchk( hipMalloc( (void **)&gpu_triangle_array, sizeof(CompFab::Triangle) * triangles.size() ) );
	gpuErrchk( hipMemcpy( gpu_triangle_array, triangle_array, sizeof(CompFab::Triangle) * triangles.size(), hipMemcpyHostToDevice ) );

	float3 lower_left = make_float3(g_voxelGrid->m_lowerLeft.m_x, g_voxelGrid->m_lowerLeft.m_y, g_voxelGrid->m_lowerLeft.m_z);
	
	vox3D voxelizer(gpu_inside_array, w, h, d, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, double_thick, lower_left, samples);
	
	voxelize_kernel_3D <<< Dg, Db>>> (voxelizer, devStates);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMemcpy( g_voxelGrid->m_insideArray, gpu_inside_array, sizeof(bool) * w * h * d, hipMemcpyDeviceToHost ) );

	gpuErrchk( hipFree(gpu_inside_array) );
	gpuErrchk( hipFree(gpu_triangle_array) );
	
}

// --------------------------- 2D square grid -------------------------
__global__ void voxelize_kernel_2D(square2D square_grid, hiprandState* globalState)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	
	int2 Index;
	Index.x = xIndex;
	Index.y = yIndex;
	

	square_grid.voxelize_kernel(Index);
}

// voxelize the given mesh with the given resolution and dimensions
void kernel_wrapper_2D(int samples, int w, int h, CompFab::VoxelGrid *g_voxelGrid, std::vector<CompFab::Triangle> triangles, bool double_thick)
{
	
	int blocksInX = (w+8-1)/8;
	int blocksInY = (h+8-1)/8;

	dim3 Dg(blocksInX, blocksInY);
	dim3 Db(8, 8);

	hiprandState* devStates;
	if (samples > 0) {
		// set up random numbers
		dim3 tpb(RANDOM_SEEDS,1,1);
	    hipMalloc ( &devStates, RANDOM_SEEDS*sizeof( hiprandState ) );
	    // setup seeds
	    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );
	}
	
	// set up boolean array on the GPU
	bool *gpu_inside_array;
	gpuErrchk( hipMalloc( (void **)&gpu_inside_array, sizeof(bool) * w * h ) );
	gpuErrchk( hipMemcpy( gpu_inside_array, g_voxelGrid->m_insideArray, sizeof(bool) * w * h, hipMemcpyHostToDevice ) );

	// set up triangle array on the GPU
	CompFab::Triangle* triangle_array = &triangles[0];
	CompFab::Triangle* gpu_triangle_array;
	gpuErrchk( hipMalloc( (void **)&gpu_triangle_array, sizeof(CompFab::Triangle) * triangles.size() ) );
	gpuErrchk( hipMemcpy( gpu_triangle_array, triangle_array, sizeof(CompFab::Triangle) * triangles.size(), hipMemcpyHostToDevice ) );

	float2 lower_left = make_float2(g_voxelGrid->m_lowerLeft.m_x, g_voxelGrid->m_lowerLeft.m_y);
	
	square2D square_grid(gpu_inside_array, w, h, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, double_thick, lower_left, samples);
	
	voxelize_kernel_2D <<< Dg, Db>>> (square_grid, devStates);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMemcpy( g_voxelGrid->m_insideArray, gpu_inside_array, sizeof(bool) * w * h, hipMemcpyDeviceToHost ) );

	gpuErrchk( hipFree(gpu_inside_array) );
	gpuErrchk( hipFree(gpu_triangle_array) );
	
}